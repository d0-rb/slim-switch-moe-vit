#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <vector>

// CUDA runtime
#include <hip/hip_runtime.h>                                                                                             
#include <hip/hip_runtime.h>                                                                                                 
#include <hipblas.h>                                                                                                    
                                                                                                                            
// CUDA and CUBLAS functions                                                                                              
//#include <helper_functions.h>                                                                                             
#include <hip/hip_runtime_api.h> 


typedef float data_t;
size_t batch_size = 4096;
size_t top_k = 2;
size_t num_expert = 128;
size_t in_feat = 512;
size_t out_feat = 2048;

#define CEIL(_x_,_y_) (((_x_)-1)/(_y_)+1)

template <typename scalar_t>
__global__
void generate_ptr_offset_kernel(size_t n, const scalar_t* base, size_t stride, const size_t* offset, const scalar_t** ptrs) {
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < n) {
		ptrs[idx] = base + stride * offset[idx];
	}
}

inline hipblasStatus_t cublasXgemmBatched(hipblasHandle_t handle,
                                  hipblasOperation_t transa,
                                  hipblasOperation_t transb,
                                  int m, int n, int k,
                                  const float           *alpha,
                                  const float           *Aarray[], int lda,
                                  const float           *Barray[], int ldb,
                                  const float           *beta,
                                  float           *Carray[], int ldc,
                                  int batchCount)
{
    return hipblasSgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda, Barray, ldb, beta, Carray, ldc, batchCount);
}

inline hipblasStatus_t cublasXgemmBatched(hipblasHandle_t handle,
                                  hipblasOperation_t transa,
                                  hipblasOperation_t transb,
                                  int m, int n, int k,
                                  const double           *alpha,
                                  const double           *Aarray[], int lda,
                                  const double           *Barray[], int ldb,
                                  const double           *beta,
                                  double           *Carray[], int ldc,
                                  int batchCount)
{
    return hipblasDgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda, Barray, ldb, beta, Carray, ldc, batchCount);
}

inline hipblasStatus_t cublasXgemmBatched(hipblasHandle_t handle,
                                  hipblasOperation_t transa,
                                  hipblasOperation_t transb,
                                  int m, int n, int k,
                                  const __half           *alpha,
                                  const __half           *Aarray[], int lda,
                                  const __half           *Barray[], int ldb,
                                  const __half           *beta,
                                  __half           *Carray[], int ldc,
                                  int batchCount)
{
    return hipblasHgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda, Barray, ldb, beta, Carray, ldc, batchCount);
}

template <typename scalar_t>
void moe_cuda_forward_impl(
        const scalar_t* input,
        const size_t* gate,
        const scalar_t* weight,
        scalar_t* output,
        const size_t batch_size,
        const size_t top_k,
        const size_t in_feat,
        const size_t out_feat) {
    

    hipblasHandle_t handle;
	hipStream_t st;
	hipStreamCreate(&st);
    checkCudaErrors(hipblasCreate(&handle));
    checkCudaErrors(hipblasSetStream(handle, st));

    // setup Aarray, Barray and Carray
	std::vector<const scalar_t*> aptrs;
    std::vector<scalar_t*> cptrs;
	
    const scalar_t **Aarray;
    const scalar_t **Barray;
    scalar_t **Carray;
	checkCudaErrors(hipMalloc(&Aarray, batch_size * sizeof(const scalar_t*) * top_k));
    checkCudaErrors(hipMalloc(&Barray, batch_size * sizeof(const scalar_t*) * top_k));
    checkCudaErrors(hipMalloc(&Carray, batch_size * sizeof(scalar_t*) * top_k));

	for (size_t i=0; i<batch_size; ++i) {
        for (size_t k=0; k<top_k; ++k) {
            aptrs.push_back(input + in_feat * i);
            // bptrs.push_back(weight + out_feat * in_feat * gate[i * top_k + k]);
            cptrs.push_back(output + out_feat * (i * top_k + k));
        }
	}
	checkCudaErrors(hipMemcpy(Aarray, aptrs.data(), batch_size * sizeof(const scalar_t*) * top_k, hipMemcpyHostToDevice));
	// checkCudaErrors(hipMemcpy(ptrs + batch_size * top_k, bptrs.data(), batch_size * sizeof(scalar_t*) * top_k, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(Carray, cptrs.data(), batch_size * sizeof(scalar_t*) * top_k, hipMemcpyHostToDevice));

	dim3 griddim(CEIL(batch_size * top_k, 256));
	dim3 blockdim(256);
    generate_ptr_offset_kernel<<<griddim, blockdim, 0, st>>>(batch_size * top_k, weight, out_feat * in_feat, gate, Barray);

    scalar_t alpha = 1, beta = 0;
	checkCudaErrors(cublasXgemmBatched(handle, 
			HIPBLAS_OP_N,
			HIPBLAS_OP_T,
			1, out_feat, in_feat,
			&alpha,
			Aarray, 1,
			Barray, out_feat,
			&beta,
			Carray, 1,
			batch_size));

	checkCudaErrors(hipStreamSynchronize(st));
    checkCudaErrors(hipStreamDestroy(st));
    checkCudaErrors(hipblasDestroy(handle));
}


int main() {
	data_t *input, *weight;
	data_t *output;
	size_t *gate;

	checkCudaErrors(hipMalloc(&input, batch_size * in_feat * sizeof(data_t)));
	checkCudaErrors(hipMalloc(&weight, num_expert * in_feat * out_feat * sizeof(data_t)));	
	checkCudaErrors(hipMalloc(&output, batch_size * top_k * out_feat * sizeof(data_t)));
	checkCudaErrors(hipMalloc(&gate, batch_size * top_k * sizeof(size_t)));

	moe_cuda_forward_impl<data_t>(input, gate, weight, output, batch_size, top_k, in_feat, out_feat);
}