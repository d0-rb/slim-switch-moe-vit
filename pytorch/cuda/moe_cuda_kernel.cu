#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <torch/torch.h>
#include <cstdio>
#include <iostream>
#include <vector>
#include <cassert>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>                                                                                          
#include <hip/hip_runtime_api.h> 

// #include "timer.hh"

#define CEIL(_x_,_y_) (((_x_)-1)/(_y_)+1)


class Helper {
public:
    Helper(const size_t num_expert_) : num_expert(num_expert_) {
        streams = new hipStream_t[num_expert];
        checkCudaErrors(hipblasCreate(&handle));
        for (size_t i=0; i<num_expert; ++i) {
            checkCudaErrors(hipStreamCreate(streams+i));
        }
    }
    ~Helper() {
        for (size_t i=0; i<num_expert; ++i) {
            checkCudaErrors(hipStreamDestroy(*(streams+i)));
        }
        checkCudaErrors(hipblasDestroy(handle));
    }
    const size_t num_expert;
    hipblasHandle_t handle;
    hipStream_t* streams;
}; 

Helper* helper = NULL;
Helper* getHelper(const size_t num_expert) { 
    if (!helper) {
        helper = new Helper(num_expert);        
    }
    assert(helper->num_expert == num_expert);
    return helper;
}


template <typename scalar_t>
__global__
void generate_ptr_offset_kernel(size_t n, const scalar_t* base, size_t stride, const int* offset, const scalar_t** ptrs) {
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < n) {
		ptrs[idx] = base + stride * offset[idx];
	}
}


inline hipblasStatus_t cublasXgemmBatched(hipblasHandle_t handle,
                                  hipblasOperation_t transa,
                                  hipblasOperation_t transb,
                                  int m, int n, int k,
                                  const float           *alpha,
                                  const float           *Aarray[], int lda,
                                  const float           *Barray[], int ldb,
                                  const float           *beta,
                                  float           *Carray[], int ldc,
                                  int batchCount) {
    return hipblasSgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda, Barray, ldb, beta, Carray, ldc, batchCount);
}

inline hipblasStatus_t cublasXgemmBatched(hipblasHandle_t handle,
                                  hipblasOperation_t transa,
                                  hipblasOperation_t transb,
                                  int m, int n, int k,
                                  const double           *alpha,
                                  const double           *Aarray[], int lda,
                                  const double           *Barray[], int ldb,
                                  const double           *beta,
                                  double           *Carray[], int ldc,
                                  int batchCount) {
    return hipblasDgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda, Barray, ldb, beta, Carray, ldc, batchCount);
}

inline hipblasStatus_t cublasXgemmBatched(hipblasHandle_t handle,
                                  hipblasOperation_t transa,
                                  hipblasOperation_t transb,
                                  int m, int n, int k,
                                  const __half           *alpha,
                                  const __half           *Aarray[], int lda,
                                  const __half           *Barray[], int ldb,
                                  const __half           *beta,
                                  __half           *Carray[], int ldc,
                                  int batchCount) {
    return hipblasHgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda, Barray, ldb, beta, Carray, ldc, batchCount);
}


inline hipblasStatus_t cublasXgemm(hipblasHandle_t handle,
                                hipblasOperation_t transa, hipblasOperation_t transb,
                                int m, int n, int k,
                                const float           *alpha,
                                const float           *A, int lda,
                                const float           *B, int ldb,
                                const float           *beta,
                                float           *C, int ldc) {
    return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

inline hipblasStatus_t cublasXgemm(hipblasHandle_t handle,
                                hipblasOperation_t transa, hipblasOperation_t transb,
                                int m, int n, int k,
                                const double          *alpha,
                                const double          *A, int lda,
                                const double          *B, int ldb,
                                const double          *beta,
                                double          *C, int ldc) {
    return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

inline hipblasStatus_t cublasXgemm(hipblasHandle_t handle,
                                hipblasOperation_t transa, hipblasOperation_t transb,
                                int m, int n, int k,
                                const __half *alpha,
                                const __half *A, int lda,
                                const __half *B, int ldb,
                                const __half *beta,
                                __half *C, int ldc) {
    return hipblasHgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

template <typename scalar_t>
void moe_cuda_forward_impl(
        const scalar_t* input,
        const int* gate,
        const scalar_t* weight,
        scalar_t* output,
        const size_t batch_size,
        const size_t in_feat,
        const size_t out_feat,
        const size_t num_expert,
        hipblasOperation_t transb) {

    Helper* h = getHelper(num_expert);

    checkCudaErrors(hipblasSetStream(h->handle, *(h->streams)));

    // setup Aarray, Barray and Carray
	std::vector<const scalar_t*> aptrs;
    std::vector<scalar_t*> cptrs;
	
    const scalar_t **Aarray;
    const scalar_t **Barray;
    scalar_t **Carray;
	checkCudaErrors(hipMalloc(&Aarray, batch_size * sizeof(const scalar_t*)));
    checkCudaErrors(hipMalloc(&Barray, batch_size * sizeof(const scalar_t*)));
    checkCudaErrors(hipMalloc(&Carray, batch_size * sizeof(scalar_t*)));

	for (size_t i=0; i<batch_size; ++i) {
        aptrs.push_back(input + in_feat * i);
        cptrs.push_back(output + out_feat * i);
	}
	checkCudaErrors(hipMemcpy(Aarray, aptrs.data(), batch_size * sizeof(const scalar_t*), hipMemcpyHostToDevice));
	// checkCudaErrors(hipMemcpy(ptrs + batch_size * top_k, bptrs.data(), batch_size * sizeof(scalar_t*) * top_k, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(Carray, cptrs.data(), batch_size * sizeof(scalar_t*), hipMemcpyHostToDevice));

	dim3 griddim(CEIL(batch_size, 256));
	dim3 blockdim(256);
    generate_ptr_offset_kernel<<<griddim, blockdim, 0, *(h->streams)>>>(batch_size, weight, out_feat * in_feat, gate, Barray);

    scalar_t alpha = 1, beta = 0;
	checkCudaErrors(cublasXgemmBatched(h->handle, 
			HIPBLAS_OP_N,
			transb,
			1, out_feat, in_feat,
			&alpha,
			Aarray, 1,
			Barray, (transb == HIPBLAS_OP_T) ? out_feat : in_feat,
			&beta,
			Carray, 1,
			batch_size));

	checkCudaErrors(hipStreamSynchronize(*(h->streams)));
}

template <typename scalar_t>
void moe_cuda_grad_weight(
        const scalar_t* input,
        const int* gate,
        const scalar_t* grad_output,
        scalar_t* grad_weight, // [num_expert x out_feat x in_feat]
        const size_t batch_size,
        const size_t in_feat,
        const size_t out_feat,
        const size_t num_expert) {

    Helper* h = getHelper(num_expert);
    
    int* gate_host = new int[batch_size];
    scalar_t alpha = 1, beta = 1;
    checkCudaErrors(hipMemcpy(gate_host, gate, batch_size * sizeof(int), hipMemcpyDeviceToHost));
    for (size_t i=0; i<batch_size; ++i) {
        checkCudaErrors(hipblasSetStream(h->handle, *(h->streams + gate_host[i])));
        checkCudaErrors(cublasXgemm(h->handle,
            HIPBLAS_OP_N, 
            HIPBLAS_OP_T,
            out_feat, 
            in_feat, 
            1,
            &alpha,
            grad_output + i * out_feat,
            out_feat,
            input + i * in_feat,
            in_feat,
            &beta,
            grad_weight + gate_host[i] * out_feat * in_feat,
            out_feat));
    }
    for (size_t i=0; i<num_expert; ++i) {
        checkCudaErrors(hipStreamSynchronize(*(h->streams + i)));
    }
    delete[] gate_host;
}

std::vector<torch::Tensor> moe_cuda_forward(
        torch::Tensor input,
        torch::Tensor gate,
        torch::Tensor weight) {
    const auto batch_size = input.size(0);
    const auto num_expert = weight.size(0);
    const auto out_feat = weight.size(1);
    const auto in_feat = weight.size(2);
            
    printf("[forward] b=%ld, expert=%ld, in_feat (d_model)=%ld, out_feat (d_ffn)=%ld\n", batch_size, num_expert, in_feat, out_feat);
    auto output = input.new_zeros({batch_size, out_feat});
    
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_forward_cuda", ([&] {
                moe_cuda_forward_impl<scalar_t>(
                    input.data_ptr<scalar_t>(),
                    gate.data_ptr<int>(),
                    weight.data_ptr<scalar_t>(),
                    output.data_ptr<scalar_t>(),
                    batch_size,
                    in_feat,
                    out_feat,
                    num_expert,
                    HIPBLAS_OP_T
                );
    }));
    
    return {output, };           
}

std::vector<torch::Tensor> moe_cuda_backward(
    torch::Tensor grad_output, // [batch_size x out_feat]
    torch::Tensor input, // [batch_size x out_feat]
    torch::Tensor gate,  // [batch_size]
    torch::Tensor weight // [num_expert x out_feat x in_feat]
) {
    const auto batch_size = input.size(0);
    const auto num_expert = weight.size(0);
    const auto out_feat = weight.size(1);
    const auto in_feat = weight.size(2);
    printf("[backward] b=%ld, expert=%ld, in_feat (d_model)=%ld, out_feat (d_ffn)=%ld\n", batch_size, num_expert, in_feat, out_feat);

    auto grad_input = grad_output.new_zeros({batch_size, in_feat});  // batch_size x in_feat
    auto grad_weight = grad_output.new_zeros({num_expert, out_feat, in_feat}); // num_expert x out_feat x in_feat

    // grad_input is easy to compute, exactly the same as forward
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_cuda_backward", ([&] {
        moe_cuda_forward_impl<scalar_t>(
            grad_output.data_ptr<scalar_t>(),
            gate.data_ptr<int>(),
            weight.data_ptr<scalar_t>(),
            grad_input.data_ptr<scalar_t>(),
            batch_size,
            out_feat,
            in_feat,
            num_expert,
            HIPBLAS_OP_N
        );
    }));

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_cuda_backward", ([&] {
        moe_cuda_grad_weight<scalar_t>(
            input.data_ptr<scalar_t>(),
            gate.data_ptr<int>(),
            grad_output.data_ptr<scalar_t>(),
            grad_weight.data_ptr<scalar_t>(),
            batch_size,
            in_feat,
            out_feat,
            num_expert
        );
    }));

    return {grad_input, grad_weight};
}


/*
int main() {
    typedef float data_t;
    size_t batch_size = 4096;
    size_t top_k = 2;
    size_t num_expert = 128;
    size_t in_feat = 1024;
    size_t out_feat = 4096;
	data_t *input, *weight;
	data_t *output;
	size_t *gate;

	checkCudaErrors(hipMalloc(&input, batch_size * in_feat * sizeof(data_t)));
	checkCudaErrors(hipMalloc(&weight, num_expert * in_feat * out_feat * sizeof(data_t)));	
	checkCudaErrors(hipMalloc(&output, batch_size * top_k * out_feat * sizeof(data_t)));
    checkCudaErrors(hipMalloc(&gate, batch_size * top_k * sizeof(size_t)));
    
    size_t nt = 16;
    double tsum = 0, tmax = 0;

    size_t *gate_host = new size_t[batch_size * top_k];
    for (size_t i=0; i<batch_size * top_k; ++i) {
        gate_host[i] = rand() % num_expert;
    } 
    checkCudaErrors(hipMemcpy(gate, gate_host, batch_size * top_k * sizeof(size_t), hipMemcpyHostToDevice));

    moe_first_linear_cuda_forward<data_t>(input, gate, weight, output, batch_size, top_k, in_feat, out_feat);
    
    for (size_t i=0; i<nt; ++i) {
        timestamp(start);
		moe_first_linear_cuda_forward<data_t>(input, gate, weight, output, batch_size, top_k, in_feat, out_feat);
		timestamp(end);
		auto t = getDuration(start, end);
		tsum += t;
		if (t > tmax) tmax = t;
    }
    printf("Mean %.3lf us, max %.3lf us\n", tsum / nt * 1e6, tmax * 1e6);
	double tflops = (double)batch_size * top_k * in_feat * out_feat * nt * 2e-12 / tsum;
	printf("%.3lf TFLOPs\n", tflops);
}
*/