#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <torch/torch.h>
#include <cstdio>
#include <iostream>
#include <vector>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>                                                                                          
#include <hip/hip_runtime_api.h> 
#include <c10/cuda/CUDAGuard.h>

#include "timer.hh"

#include "cublas_wrapper.h"
#include "cuda_stream_manager.h"

#define CEIL(_x_,_y_) (((_x_)-1)/(_y_)+1)

// #define MOE_BREAKDOWN
// #define MOE_DEBUG

thread_local CudaStreamManager smgr;

template <typename scalar_t>
__global__
void generate_ptr_offset_kernel(size_t n, const scalar_t* base, size_t stride,
		const int* offset, const scalar_t** ptrs) { 
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < n) {
		ptrs[idx] = base + stride * offset[idx];
	}
}


template <typename scalar_t>
__global__
void batch_scatter_kernel(size_t wid, const int* pos, 
		const scalar_t* inbuf, scalar_t* oubuf) { 
	inbuf += wid * blockIdx.x;
	oubuf += wid * pos[blockIdx.x];
	for (int i = threadIdx.x; i < wid; i += blockDim.x) {
		oubuf[i] = inbuf[i];
	}
}

void moe_cuda_expert_count_impl(
        const int* d_gate,
		int* expert_count,
		int* d_pos,
		const size_t num_expert,
        const size_t batch_size) {
    int *gate = new int[batch_size];
	int *expert_ptr = new int[num_expert];
	memset(expert_count, 0, sizeof(int) * num_expert);

	checkCudaErrors(hipMemcpy(gate, d_gate, sizeof(int) * batch_size,
				hipMemcpyDeviceToHost));

	for (int i = 0; i < batch_size; ++i) {
		++expert_count[gate[i]];
	}
	expert_ptr[0] = 0;
	for (int i = 1; i < num_expert; ++i) {
		expert_ptr[i] = expert_ptr[i - 1] + expert_count[i - 1];
	}

	int *pos = new int[batch_size];

	for (int i = 0; i < batch_size; ++i) {
		pos[i] = expert_ptr[gate[i]]++;
	}
	checkCudaErrors(hipMemcpy(d_pos, pos, sizeof(int) * batch_size,
				hipMemcpyHostToDevice));
	delete [] gate;
	delete [] expert_ptr;

	ENSURE_SMGR(smgr, num_expert);
}

template <typename scalar_t>
void moe_cuda_local_scatter_impl(
        const scalar_t* input,
		const int* d_pos,
		scalar_t* input_buf,
		const size_t batch_size,
		const size_t in_feat) {
	batch_scatter_kernel<scalar_t>
		<<<batch_size, 256, 0, smgr.streams[0]>>>(in_feat, d_pos, input,
				input_buf); 
	smgr.sync(0);
}

template <typename scalar_t>
__global__
void batch_gather_kernel(size_t wid, const int* pos, 
		const scalar_t* inbuf, scalar_t* oubuf) { 
	inbuf += wid * pos[blockIdx.x];
	oubuf += wid * blockIdx.x;
	for (int i = threadIdx.x; i < wid; i += blockDim.x) {
		oubuf[i] = inbuf[i];
	}
}

template <typename scalar_t>
void moe_cuda_local_gather_impl(
        const scalar_t* output_buf,
		const int* d_pos,
		scalar_t* output,
		const size_t batch_size,
		const size_t out_feat) {
	batch_gather_kernel<scalar_t>
		<<<batch_size, 256, 0, smgr.streams[0]>>>(out_feat, d_pos, output_buf,
				output); 
	smgr.sync(0);
}

template <typename scalar_t>
void moe_cuda_forward_impl(
        const scalar_t* input_buf,
        const scalar_t* weight,
		const int* expert_count,
        scalar_t* output_buf,
        const size_t in_feat,
        const size_t out_feat,
        const size_t num_expert) {
	scalar_t alpha = 1, beta = 0; 

	for (int i = 0, ptr = 0; i < num_expert; ++i) {
		if (expert_count[i] == 0) {
			continue;
		}
		// Use T(B) x T(A) = T(C) to produce row-major C
		checkCudaErrors(cublasXgemm(smgr.handles[i],
				HIPBLAS_OP_T,
				HIPBLAS_OP_N,
				out_feat, expert_count[i], in_feat,
				&alpha,
				weight + i * in_feat * out_feat, in_feat,
				input_buf + ptr * in_feat, in_feat,
				&beta,
				output_buf + out_feat * ptr, out_feat
				));

		ptr += expert_count[i];
	}
	smgr.sync();
}

template <typename scalar_t>
void moe_cuda_backward_impl(
        const scalar_t* grad_output_buf,
        const scalar_t* input_buf,
		const scalar_t* weight,
		const int* expert_count,
        scalar_t* grad_input_buf,
        scalar_t* grad_weight,
        const size_t batch_size,
        const size_t in_feat,
        const size_t out_feat,
        const size_t num_expert) {
	ENSURE_SMGR(smgr, num_expert);
    scalar_t alpha = 1, beta = 0;

	for (int i = 0, ptr = 0; i < num_expert; ++i) {
		if (expert_count[i] == 0) {
			hipMemset(grad_weight + i * in_feat * out_feat, 0, 
					sizeof(scalar_t) * in_feat * out_feat);
			continue;
		}
		// Use T(B) x T(A) = T(C) to produce row-major C

		// Backward input: g_i = w @ g_o
		checkCudaErrors(cublasXgemm(smgr.handles[i],
				HIPBLAS_OP_N,
				HIPBLAS_OP_N,
				in_feat, expert_count[i], out_feat,
				&alpha,
				weight + i * in_feat * out_feat, in_feat,
				grad_output_buf + ptr * out_feat, out_feat,
				&beta,
				grad_input_buf + in_feat * ptr, in_feat
				));

		// Backward weight: g_w = i @ g_o
		checkCudaErrors(cublasXgemm(smgr.handles[i],
				HIPBLAS_OP_N,
				HIPBLAS_OP_T,
				in_feat, out_feat, expert_count[i],
				&alpha,
				input_buf + in_feat * ptr, in_feat,
				grad_output_buf + ptr * out_feat, out_feat,
				&beta,
				grad_weight + i * in_feat * out_feat, in_feat
				));

		ptr += expert_count[i];
	}
	smgr.sync();
}


std::vector<torch::Tensor> moe_cuda_expert_count(
		torch::Tensor weight,
		torch::Tensor gate) {
	const auto batch_size = gate.size(0);
	const auto num_expert = weight.size(0);

	auto ec_options = torch::TensorOptions().dtype(torch::kInt32);
	auto expert_count = torch::empty(num_expert, ec_options);

	auto pos_options = torch::TensorOptions()
		.device(gate.device())
		.dtype(torch::kInt32);
	auto pos = torch::empty(batch_size, pos_options);
	moe_cuda_expert_count_impl(
			gate.data_ptr<int>(),
			expert_count.data_ptr<int>(),
			pos.data_ptr<int>(),
			num_expert,
			batch_size);

	return {expert_count, pos};
}

std::vector<torch::Tensor> moe_cuda_local_scatter(
    torch::Tensor input,
	torch::Tensor pos) {
	const auto batch_size = input.size(0);
    const auto in_feat = input.size(1);

	auto input_buf = torch::empty_like(input);

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_local_scatter_cuda", 
			([&] {
		moe_cuda_local_scatter_impl<scalar_t>(
			input.data_ptr<scalar_t>(),
			pos.data_ptr<int>(),
			input_buf.data_ptr<scalar_t>(),
			batch_size,
			in_feat);
	}));
	return {input_buf,};
}

std::vector<torch::Tensor> moe_cuda_local_gather(
	torch::Tensor output_buf,
	torch::Tensor pos) {
	const auto batch_size = output_buf.size(0);
    const auto out_feat = output_buf.size(1);

	auto output = torch::empty_like(output_buf);

    AT_DISPATCH_FLOATING_TYPES(output_buf.scalar_type(), "moe_local_gather_cuda", 
			([&] {
		moe_cuda_local_gather_impl<scalar_t>(
			output_buf.data_ptr<scalar_t>(),
			pos.data_ptr<int>(),
			output.data_ptr<scalar_t>(),
			batch_size,
			out_feat);
	}));
	return {output,};
}

std::vector<torch::Tensor> moe_cuda_forward(
        torch::Tensor input_buf,
        torch::Tensor weight,
		torch::Tensor expert_count
		) {
	const auto batch_size = input_buf.size(0);
    const auto num_expert = weight.size(0);
    const auto out_feat = weight.size(1);
    const auto in_feat = weight.size(2);
            
#ifdef MOE_DEBUG
    printf("[forward] expert=%ld, in_feat (d_model)=%ld, out_feat (d_ffn)=%ld\n", 
			num_expert, in_feat, out_feat);
#endif
	/*
    const int device = device_of(input).value().index();
    if (smgr.streams == NULL) {
        smgr.setup(num_expert, device);
    }
	*/
	auto out_options = torch::TensorOptions()
		.device(input_buf.device())
		.dtype(input_buf.dtype());
    auto output = torch::empty({batch_size, out_feat}, out_options);
    
    AT_DISPATCH_FLOATING_TYPES(input_buf.scalar_type(), "moe_forward_cuda", 
			([&] {
		moe_cuda_forward_impl<scalar_t>(
			input_buf.data_ptr<scalar_t>(),
			weight.data_ptr<scalar_t>(),
			expert_count.data_ptr<int>(),
			output.data_ptr<scalar_t>(),
			in_feat,
			out_feat,
			num_expert
		);
    }));
    
    return {output, };           
}

std::vector<torch::Tensor> moe_cuda_backward(
    torch::Tensor grad_output_buf, // [batch_size x out_feat]
    torch::Tensor input_buf, // [batch_size x out_feat]
    torch::Tensor weight, // [num_expert x out_feat x in_feat]
	torch::Tensor expert_count
) {
    const auto batch_size = input_buf.size(0);
    const auto num_expert = weight.size(0);
    const auto out_feat = weight.size(1);
    const auto in_feat = weight.size(2);

#ifdef MOE_DEBUG
    printf("[backward] b=%ld, expert=%ld, in_feat (d_model)=%ld, "
			"out_feat (d_ffn)=%ld\n",
			batch_size, num_expert, in_feat, out_feat);
#endif

    auto grad_input_buf = grad_output_buf.new_empty({batch_size, in_feat}); 
    auto grad_weight = grad_output_buf.new_empty({num_expert, out_feat, in_feat});

    AT_DISPATCH_FLOATING_TYPES(input_buf.scalar_type(), "moe_cuda_backward", ([&] {
        moe_cuda_backward_impl<scalar_t>(
            grad_output_buf.data_ptr<scalar_t>(),
            input_buf.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
			expert_count.data_ptr<int>(),
            grad_input_buf.data_ptr<scalar_t>(),
            grad_weight.data_ptr<scalar_t>(),
            batch_size,
            in_feat,
            out_feat,
            num_expert
        );
    }));

    return {grad_input_buf, grad_weight};
}


/*
int main() {
    typedef float data_t;
    size_t batch_size = 4096;
    size_t top_k = 2;
    size_t num_expert = 128;
    size_t in_feat = 1024;
    size_t out_feat = 4096;
	data_t *input, *weight;
	data_t *output;
	size_t *gate;

	checkCudaErrors(hipMalloc(&input, batch_size * in_feat * sizeof(data_t)));
	checkCudaErrors(hipMalloc(&weight, num_expert * in_feat * out_feat * sizeof(data_t)));	
	checkCudaErrors(hipMalloc(&output, batch_size * top_k * out_feat * sizeof(data_t)));
    checkCudaErrors(hipMalloc(&gate, batch_size * top_k * sizeof(size_t)));
    
    size_t nt = 16;
    double tsum = 0, tmax = 0;

    size_t *gate_host = new size_t[batch_size * top_k];
    for (size_t i=0; i<batch_size * top_k; ++i) {
        gate_host[i] = rand() % num_expert;
    } 
    checkCudaErrors(hipMemcpy(gate, gate_host, batch_size * top_k * sizeof(size_t), hipMemcpyHostToDevice));

    moe_first_linear_cuda_forward<data_t>(input, gate, weight, output, batch_size, top_k, in_feat, out_feat);
    
    for (size_t i=0; i<nt; ++i) {
        timestamp(start);
		moe_first_linear_cuda_forward<data_t>(input, gate, weight, output, batch_size, top_k, in_feat, out_feat);
		timestamp(end);
		auto t = getDuration(start, end);
		tsum += t;
		if (t > tmax) tmax = t;
    }
    printf("Mean %.3lf us, max %.3lf us\n", tsum / nt * 1e6, tmax * 1e6);
	double tflops = (double)batch_size * top_k * in_feat * out_feat * nt * 2e-12 / tsum;
	printf("%.3lf TFLOPs\n", tflops);
}
*/
