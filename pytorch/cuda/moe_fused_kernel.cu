#include "moe_cuda_kernel.h"

#include <cstdio>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h> 
#include <c10/cuda/CUDAGuard.h>

#include "cuda_stream_manager.h"

#ifdef MOE_USE_NCCL
#include <mpi.h>
#include <nccl.h>

// TODO

#endif

